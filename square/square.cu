#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void square(float* d_out, float* d_in) {
    int idx = threadIdx.x;
    float f = d_in[idx];
    d_out[idx] = f * f * f;
}

int main (int argc, char** argv) {
    int nDevices;

    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printf("Device Number: %d\n", i);
        printf("  Device name: %s\n", prop.name);
        printf("  Memory Clock Rate (MHz): %d\n",
            prop.memoryClockRate / 1024);
        printf("  Memory Bus Width (bits): %d\n",
            prop.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s): %f\n",
            2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
        printf("  Global Memory (MB): %lu\n\n",
            prop.totalGlobalMem / (1024 * 1024));
    }

    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
    
    // generate the input array on the host
    float h_in[ARRAY_SIZE];
    for (int i = 0; i < ARRAY_SIZE; i++) {   
        h_in[i] = float(i);
    }
    float h_out[ARRAY_SIZE];

    // declare GPU memory pointers
    float* d_in;
    float* d_out;

    // allocate GPU memory
    hipMalloc((void**) &d_in, ARRAY_BYTES);
    hipMalloc((void**) &d_out, ARRAY_BYTES);

    // transfer the array to the GPU
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // launch the kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back the result array to the CPU
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // print out the resulting array
    for (int i = 0; i < ARRAY_SIZE; i++) {
        printf("%f", h_out[i]);
        printf(((i % 4) != 3) ? "\t" : "\n");
    }

    // free GPU memory allocation
    hipFree(d_in);
    hipFree(d_out);
}